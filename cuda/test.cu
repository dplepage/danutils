#include "hip/hip_runtime.h"
#define W {{ W }}
#define H {{ H }}
#define N {{ W*H }}
#define PI {{ scipy.pi }}

// The following jinja2 line registers the texture with the module and emits:
// texture <float4, 2> normal;
{{ reg.texture('normal', 'float4', (N1, N2)) }}
{{ reg.texture('mask', 'unsigned char', (N1, N2)) }}
{{ reg.texture('weights', 'float', (N1, N2, 2)) }}

// The following line expands to
// float[N1*N2] data;
{{ reg.array('data', 'float', (N1, N2)) }}

{{
reg.function(
    name = "setone",
    sig = "P",
    block = (512, 1, 1),
    grid = (ceil(N1*N2/512), 1),
    texnames = ['mask'],
    doc = ("Set every entry in the array to 1"
           ""
           "Entries blocked by the mask will be ignored.")
)
}}
__global__ void setone(float* target) {
    int pixel = (blockIdx.x + gridDim.x*blockIdx.y) * blockDim.x + threadIdx.x;
    int j = pixel%W;
    int i = pixel/W;
    if (i < H) {
        if (maskI(j,i) != 0) {
            outI(i,j) = 1;
        } else {
            outI(i,j) = 0;
        }
    }
}
